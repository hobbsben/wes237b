#include "hip/hip_runtime.h"
#include "filter.h"
#include "cuda_timer.h"

__global__
void kernel_filter(const uchar * input, uchar * output, const uint height, const uint width)
{
	// TODO: Implement a blur filter for the camera (averaging an NxN array of pixels
 //printf("%d\n", threadIdx.x);
//****************************************************************
 int idx = blockIdx.x * blockDim.x + threadIdx.x;
 int idy = blockIdx.y * blockDim.y + threadIdx.y;
  if (idx > width -3 || idy > height-3 || idx < 2 || idy < 2) {
      return;
  }
  
  float c = 0.0f;

// printf("/n/n output: %0.2c", output[idx+idy]; 

	for (int i = -2; i <= 2; i++) 
	{
		for (int j = -2; j <= 2; j++)
		{
     		 c += input[(idy+i)*width +idx+j];
               
   		 }
		
  	}
if(c > 255*25){c=6375;} 
output[(idy)*width +idx] =c/25.0f;    //calculate the average by dividing by 0

// check to see if pixel is greater than 0
if(c/25 > 255 || c/25 < 0)
	{	
	printf(" %.2f \n",c/25);
	}
//****************************************************************

}//end kernel_filter 
inline int divup(int a, int b)
{
	if (a % b)
		return a / b + 1;
	else
		return a / b;
}

/**
 * Wrapper for calling the kernel.
 */
void filter_gpu(const uchar * input, uchar * output, const uint height, const uint width)
{
       // printf("\n\n testing filter_gpu function /n/n");

	CudaSynchronizedTimer timer;

	// Launch the kernel
	const int grid_x = 64;
	const int grid_y = 64;
     
        int block_x = divup(width, grid_x); // calculate the grid size
        int block_y = divup(height, grid_y); // calculate the grid size
        
	// dim3 grid(1, 1, 1);  // TODO
	// dim3 block(1, 1, 1); // TODO
        dim3 grid(grid_x, grid_y, 1);  // TODO
	dim3 block(block_x, block_y, 1); // TODO

	timer.start();
        //printf("\n\n testing filter_gpu function /n/n");
	kernel_filter<<<grid, block>>>(input, output, height, width);
	timer.stop();
//        printf("\n\n testing filter_gpu function /n/n");
	hipDeviceSynchronize();

	float time_kernel = timer.getElapsed();
}





