#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "matrixmul.h"
#include "cuda_error.h"

#define BLOCK_SIZE 16

__global__ void block_mm_kernel(const float* A, const float* B, float* output, int M, int N) 
{
   // Block index
            int bidx = blockIdx.x;
            int bidy = blockIdx.y;

            int tidx = threadIdx.x;
            int tidy = threadIdx.y;
            __shared__ float A_shared[BLOCK_SIZE][BLOCK_SIZE];
            __shared__ float B_shared[BLOCK_SIZE][BLOCK_SIZE];

       int aBegin = M * BLOCK_SIZE * bidy;
       int aEnd   = aBegin + M - 1;
       int aStep  = BLOCK_SIZE;

       int bBegin = BLOCK_SIZE * bidx;
       int bStep  = BLOCK_SIZE * N;

       float sum = 0;
        // loop through blocks
       for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) 
	{
A_shared[tidy][tidx] = A[M*tidy+tidx +a];                                                                 
 B_shared[tidy][tidx]= B[N*tidy+tidx +b];                                                                 
    // sync the threads
        __syncthreads();                                                                                  
//accumulation
            for (int k=0;k< BLOCK_SIZE;++k)                                                              
 sum+=A_shared[tidy][k]*B_shared[k][tidx];                                                                
                                                                                                             
// sync it
       __syncthreads();                                                                                  
        }                                                                                                     
        int c = N*BLOCK_SIZE* bidy + BLOCK_SIZE * bidx;                                                       
        output[ N * tidy + tidx +c] = sum;             
}//endline

inline int divup(int a, int b)
{
	if (a % b)
		return a / b + 1;
	else
		return a / b;
}

float run_mm_gpu(const float* A, const float* B, float* C, int M, int N)
{
	// Profiling
	float time_gpu;

	hipEvent_t start, stop;
	CudaSafeCall(hipEventCreate(&start));
	CudaSafeCall(hipEventCreate(&stop));

	CudaSafeCall(hipEventRecord(start, 0));
	
	dim3 dimGrid(divup(N, BLOCK_SIZE), divup(N, BLOCK_SIZE));
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

	// Launch kernel 
	block_mm_kernel<<<dimGrid, dimBlock>>>(A, B, C, M, N);

	CudaCheckError();

	CudaSafeCall(hipDeviceSynchronize());

	CudaSafeCall(hipEventRecord(stop, 0));
	CudaSafeCall(hipEventSynchronize(stop));
	
	CudaSafeCall(hipEventElapsedTime(&time_gpu, start, stop));

	// Free Memory
	CudaSafeCall(hipEventDestroy(start));
	CudaSafeCall(hipEventDestroy(stop));

	return time_gpu;
}


