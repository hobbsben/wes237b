#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "matrixmul.h"
#include "cuda_error.h"

#define BLOCK_SIZE 16

__global__ void block_mm_kernel(const float* A, const float* B, float* output, int M, int N) 
{
	int gidx = blockIdx.x * blockDim.x + threadIdx.x;
	int gidy = blockIdx.y * blockDim.y + threadIdx.y;
	int tidx = threadIdx.x;
        int tidy = threadIdx.y;
        float sum = 0.0;        

        const int NUM_BLOCKS = (N*M)/BLOCK_SIZE;  // calculate amount of blocks
        
        __shared__  float A_shared[BLOCK_SIZE*BLOCK_SIZE];
        __shared__  float B_shared[BLOCK_SIZE*BLOCK_SIZE];
        __shared__  float C_shared[BLOCK_SIZE*BLOCK_SIZE];

        //store input matrices into shared memory
        for(int h = 0; h < M; h+=BLOCK_SIZE)
        { 
            A_shared[tidx+tidy*BLOCK_SIZE + h] = A[tidx+tidy*BLOCK_SIZE + h];
            B_shared[tidx+tidy*BLOCK_SIZE + h] = B[tidx+tidy*BLOCK_SIZE + h];
            __syncthreads();
        

        // calculate sum
           for(int i = 0; i < NUM_BLOCKS; i++)
           {
	       //if(tidx < BLOCK_SIZE || tidy < BLOCK_SIZE)
	      // {
	           C_shared[tidx+tidy] += A_shared[gidx+gidy*M] * B_shared[gidx*N+gidy];
	      // }
           }
           output[gidx+gidy] = C_shared[tidx+tidy];
        }
        
        
}//endline


inline int divup(int a, int b)
{
	if (a % b)
		return a / b + 1;
	else
		return a / b;
}


float run_mm_gpu(const float* A, const float* B, float* C, int M, int N)
{
	// Profiling
	float time_gpu;

	hipEvent_t start, stop;
	CudaSafeCall(hipEventCreate(&start));
	CudaSafeCall(hipEventCreate(&stop));

	CudaSafeCall(hipEventRecord(start, 0));
	
	dim3 dimGrid(divup(N, BLOCK_SIZE), divup(N, BLOCK_SIZE));
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

	// Launch kernel 
	block_mm_kernel<<<dimGrid, dimBlock>>>(A, B, C, M, N);

	CudaCheckError();

	CudaSafeCall(hipDeviceSynchronize());

	CudaSafeCall(hipEventRecord(stop, 0));
	CudaSafeCall(hipEventSynchronize(stop));
	
	CudaSafeCall(hipEventElapsedTime(&time_gpu, start, stop));

	// Free Memory
	CudaSafeCall(hipEventDestroy(start));
	CudaSafeCall(hipEventDestroy(stop));

	return time_gpu;
}


