#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "matrixmul.h"
#include "cuda_error.h"

#define BLOCK_SIZE 16

__global__ void block_mm_kernel(const float* A, const float* B, float* output, int M, int N) 
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
int idy = blockIdx.y * blockDim.y + threadIdx.y;
	if(1)
	{
	output[idy*M+idx]=A[idy*M+idx]*B[idy*M+idx];
	}
}//endline


inline int divup(int a, int b)
{
	if (a % b)
		return a / b + 1;
	else
		return a / b;
}


float run_mm_gpu(const float* A, const float* B, float* C, int M, int N)
{
	// Profiling
	float time_gpu;

	hipEvent_t start, stop;
	CudaSafeCall(hipEventCreate(&start));
	CudaSafeCall(hipEventCreate(&stop));

	CudaSafeCall(hipEventRecord(start, 0));
	
	dim3 dimGrid(divup(N, BLOCK_SIZE), divup(N, BLOCK_SIZE));
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

	// Launch kernel 
	block_mm_kernel<<<dimGrid, dimBlock>>>(A, B, C, M, N);

	CudaCheckError();

	CudaSafeCall(hipDeviceSynchronize());

	CudaSafeCall(hipEventRecord(stop, 0));
	CudaSafeCall(hipEventSynchronize(stop));
	
	CudaSafeCall(hipEventElapsedTime(&time_gpu, start, stop));

	// Free Memory
	CudaSafeCall(hipEventDestroy(start));
	CudaSafeCall(hipEventDestroy(stop));

	return time_gpu;
}


