#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "matrixmul.h"
#include "cuda_error.h"

#define BLOCK_SIZE 16

__global__ void block_mm_kernel(const float* A, const float* B, float* output, int M, int N) 
{
	int gidx = blockIdx.x * blockDim.x + threadIdx.x;
	int gidy = blockIdx.y * blockDim.y + threadIdx.y;
	int tidx = threadIdx.x;
        int tidy = threadIdx.y;
        float sum = 0;
;        

        const int NUM_BLOCKS = M/(BLOCK_SIZE);  // calculate amount of blocks
        
        __shared__  float A_shared[BLOCK_SIZE*BLOCK_SIZE];
        __shared__  float B_shared[BLOCK_SIZE*BLOCK_SIZE];
        //__shared__  float C_shared[BLOCK_SIZE*BLOCK_SIZE];

        //loop through blocks in the grid 
        for(int grid_block = 0; grid_block < NUM_BLOCKS; grid_block++)       
        {   
            //check grid boundary conditions
            //if(tidx+tidy*grid_block < M*N)
            //{
                //store input matrices into shared memory
                A_shared[tidx+tidy*BLOCK_SIZE] = A[(grid_block*BLOCK_SIZE)+tidx+(tidy+gidy)*M];
                B_shared[tidx+tidy*BLOCK_SIZE] = B[gidx+tidx+(grid_block*BLOCK_SIZE+tidy)*M];
                __syncthreads();
             //}
         // loop through elements within the block
         for(int block_element = 0; block_element < BLOCK_SIZE; block_element++)
            {
               // check block boundary conditions
	       //  if(tidx+tidy*+grid_block*BLOCK_SIZE < M*N)
	       // {
	           sum += A_shared[tidx+tidy*BLOCK_SIZE] * B_shared[tidx+tidy*BLOCK_SIZE + block_element];
	       // }
           }
           output[gidx+gidy*N ] += sum;
        }
        
        
}//endline


inline int divup(int a, int b)
{
	if (a % b)
		return a / b + 1;
	else
		return a / b;
}


float run_mm_gpu(const float* A, const float* B, float* C, int M, int N)
{
	// Profiling
	float time_gpu;

	hipEvent_t start, stop;
	CudaSafeCall(hipEventCreate(&start));
	CudaSafeCall(hipEventCreate(&stop));

	CudaSafeCall(hipEventRecord(start, 0));
	
	dim3 dimGrid(divup(N, BLOCK_SIZE), divup(N, BLOCK_SIZE));
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

	// Launch kernel 
	block_mm_kernel<<<dimGrid, dimBlock>>>(A, B, C, M, N);

	CudaCheckError();

	CudaSafeCall(hipDeviceSynchronize());

	CudaSafeCall(hipEventRecord(stop, 0));
	CudaSafeCall(hipEventSynchronize(stop));
	
	CudaSafeCall(hipEventElapsedTime(&time_gpu, start, stop));

	// Free Memory
	CudaSafeCall(hipEventDestroy(start));
	CudaSafeCall(hipEventDestroy(stop));

	return time_gpu;
}


