#include "hip/hip_runtime.h"
#include "filter.h"
#include "cuda_timer.h"
#include "math.h"
#include <iostream>

using namespace std;


__global__
void kernel_sobel_filter(const uchar * input, uchar * output, const uint height, const uint width)
{ 

        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        int idy = blockIdx.y * blockDim.y + threadIdx.y;

	const int sobel_x[3][3] = {
		{-1, 0, 1},
		{-2, 0, 2},
		{-1, 0, 1}
	};
	const int sobel_y[3][3]  = {
		{-1, -2, -1},
		{0,   0,  0},
		{1,   2,  1}
	};


	// TODO

	   float sumx = 0.0f;
           float sumy = 0.0f;
	   for(int i = 0; i < 3; i++)
	   {   
                for (int j = 0; j < 3; j++)
		{
		      sumx += sobel_x[i][j] * input[(idy+i)*width +idx+j];
                      sumy += sobel_y[i][j] * input[(idy+i)*width +idx+j];
                       
 		}
	   }

	   output[(idy)*width +idx] = sqrt(pow(sumx,2) + pow(sumy,2));
        
}


inline int divup(int a, int b)
{
	if (a % b)
		return a / b + 1;
	else
		return a / b;
}

/**
 * Wrapper for calling the kernel.
 */
void sobel_filter_gpu(const uchar * input, uchar * output, const uint height, const uint width)
{
	const int size = height * width * sizeof(uchar);

	CudaSynchronizedTimer timer;


        // Launch the kernel
	const int grid_x = 64;
	const int grid_y = 64;
     
        int block_x = divup(width, grid_x); // calculate the grid size
        int block_y = divup(height, grid_y); // calculate the grid size
        
	// dim3 grid(1, 1, 1);  // TODO
	// dim3 block(1, 1, 1); // TODO
        dim3 grid(grid_x, grid_y, 1);  // TODO
	dim3 block(block_x, block_y, 1); // TODO
	timer.start();
	kernel_sobel_filter<<<grid, block>>>(input, output, height, width);
	timer.stop();

	hipDeviceSynchronize();

	float time_kernel = timer.getElapsed();
}


void sobel_filter_cpu(const uchar * input, uchar * output, const uint height, const uint width)
{
	const int sobel_x[3][3] = {
		{-1, 0, 1},
		{-2, 0, 2},
		{-1, 0, 1}
	};
	const int sobel_y[3][3]  = {
		{-1, -2, -1},
		{0,   0,  0},
		{1,   2,  1}
	};

	for (uint y = 1; y < height - 1; ++y)
	{
		for (uint x = 1; x < width - 1; ++x)
		{

			const int pixel_x = (int) (
					(sobel_x[0][0] * input[x-1 + (y-1) * width]) + 
					(sobel_x[0][1] * input[x   + (y-1) * width]) + 
					(sobel_x[0][2] * input[x+1 + (y-1) * width]) +
					(sobel_x[1][0] * input[x-1 + (y  ) * width]) + 
					(sobel_x[1][1] * input[x   + (y  ) * width]) + 
					(sobel_x[1][2] * input[x+1 + (y  ) * width]) +
					(sobel_x[2][0] * input[x-1 + (y+1) * width]) + 
					(sobel_x[2][1] * input[x   + (y+1) * width]) + 
					(sobel_x[2][2] * input[x+1 + (y+1) * width])
					);
			const int pixel_y = (int) (
					(sobel_y[0][0] * input[x-1 + (y-1) * width]) + 
					(sobel_y[0][1] * input[x   + (y-1) * width]) + 
					(sobel_y[0][2] * input[x+1 + (y-1) * width]) +
					(sobel_y[1][0] * input[x-1 + (y  ) * width]) + 
					(sobel_y[1][1] * input[x   + (y  ) * width]) + 
					(sobel_y[1][2] * input[x+1 + (y  ) * width]) +
					(sobel_y[2][0] * input[x-1 + (y+1) * width]) + 
					(sobel_y[2][1] * input[x   + (y+1) * width]) + 
					(sobel_y[2][2] * input[x+1 + (y+1) * width])
					);

			float magnitude = sqrt((float)(pixel_x * pixel_x + pixel_y * pixel_y));

			if (magnitude < 0){ magnitude = 0; }
			if (magnitude > 255){ magnitude = 255; }

			output[x + y * width] = magnitude;
		}
	}
}



