#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
// TODO Place your GPU kernel here

__global__  void vector_add(float *a, float *b, float *c, int)
{

	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id<size) c[id]=a[id]+b[id];
 //c[id]=a[id]+b[id];
}


/**
 * Host main routine
 */
int main(void)
{

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the host input vector A
    float *h_A = (float *)malloc(size);

    // Allocate the host input vector B
    float *h_B = (float *)malloc(size);

    // Allocate the host output vector C
    float *h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate the device input vector A
    float *d_A = NULL;
    hipMalloc(&d_A, size);// TODO
     
    // Allocate the device input vector B
    float *d_B = NULL;
    hipMalloc(&d_B, size);// TODO

    // Allocate the device output vector C
    float *d_C = NULL;
    hipMalloc(&d_C, size);	// TODO

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
	// TODO
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
   hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
 
    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid   = numElements/threadsPerBlock; // TODO
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    
    // TODO Launch kernel
    vector_add<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C);

    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
	// TODO
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED\n");

    // Free device global memory
	// TODO
        hipFree(d_A);
   	hipFree(d_B);
 	hipFree(d_C);
    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Done\n");
    return 0;
}

