#include "hip/hip_runtime.h"

#include "fir_gpu.h"
#include "cuda_timer.h"
#include <iostream>

#include <hip/hip_runtime.h>
#define BLOCK_SIZE 64


// Baseline
__global__
void fir_kernel1(const float *coeffs, const float *input, float *output, int length, int filterLength)
{
        int id = blockIdx.x * blockDim.x + threadIdx.x;
	//if(id < length-filterLength){
	// TODO

	   float sum = 0.0f;
	   for(int i = 0; i < filterLength; i++)
	   {   
		sum += coeffs[i]*input[i+id];	
 
	   }

	   output[id] = sum;
        }
//}



// Coefficients in shared memory
// Here we suppose that filterLength and BLOCK_SIZE is always 64
__global__
void fir_kernel2(const float *coeffs, const float *input, float *output, int length, int filterLength)
{
	//TODO

  int id = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__  float c[64];
	//for(int i=0; i<64;i++)
	//{c[i]=coeffs[i];}
	c[threadIdx.x] = coeffs[threadIdx.x];
	__syncthreads();
// TODO

	   float sum = 0.0f;
	   for(int i = 0; i < filterLength; ++i)
	   {   
		sum += c[i]*input[i+id];	


	   output[id] = sum;
	}
}
// Coefficients and inputs in shared memory
// Here we suppose that filterLength and BLOCK_SIZE is always 64
__global__
void fir_kernel3(const float *coeffs, const float *input, float *output, int length, int filterLength)
{
	// TODO
int id = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__  float c[64];
	c[threadIdx.x] = coeffs[threadIdx.x];
	 __shared__  float k[BLOCK_SIZE+64];
	k[threadIdx.x] = input[blockDim.x*blockIdx.x + threadIdx.x];
	k[threadIdx.x+BLOCK_SIZE] = input[blockDim.x*blockIdx.x + threadIdx.x+BLOCK_SIZE];
	__syncthreads();

	   float sum = 0.0f;
	   for(int i = 0; i < filterLength; ++i)
	   {   
		sum += c[i]*k[threadIdx.x+i];	

	        
	   }
	output[id] = sum;
}


inline int divup(int a, int b)
{
	if (a % b)
		return a / b + 1;
	else
		return a / b;
}

void fir_gpu(const float *coeffs, const float *input, float *output, int length, int filterLength)
{
	const int output_size = length - filterLength;

	CudaSynchronizedTimer timer;
        
        int grid_size = divup( length, BLOCK_SIZE); // calculate the grid size

	const int block_size = BLOCK_SIZE;

        dim3 block(block_size, 1, 1);
        dim3 grid(grid_size, 1, 1);

	timer.start();
	// TODO Launch kernel here
        //void fir_kernel1(const float *coeffs, const float *input, float *output, int length, int filterLength)
        // fir_kernel1<<<grid,block>>>(coeffs, input, output, output_size, filterLength);
        //fir_kernel2<<<grid,block>>>(coeffs, input, output, output_size, 64);
        fir_kernel3<<<grid,block>>>(coeffs, input, output, output_size, 64);
	
	timer.stop();

	hipDeviceSynchronize();

	CudaCheckError();

	float time_gpu = timer.getElapsed();
	
	//std::cout << "Kernel Time: " << time_gpu << "ms\n";
}



